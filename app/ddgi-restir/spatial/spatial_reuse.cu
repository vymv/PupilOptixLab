#include "hip/hip_runtime.h"
#include "type.h"
#include <optix.h>

//#include "../indirect/indirect.h"
#include "../indirect/probemath.h"
#include "render/geometry.h"
#include "render/emitter.h"
#include "optix/util.h"

#include "cuda/random.h"

using namespace Pupil;

extern "C" {
__constant__ ddgi::spatial::OptixLaunchParams optix_launch_params;
}

extern "C" __global__ void __raygen__main() {
    const uint3 index = optixGetLaunchIndex();
    const unsigned int w = optix_launch_params.config.frame.width;
    const unsigned int h = optix_launch_params.config.frame.height;
    const unsigned int pixel_index = index.y * w + index.x;

    float4 pos_with_flag = optix_launch_params.position_buffer[pixel_index];
    if (pos_with_flag.w <= 0.f) {
        optix_launch_params.final_reservoirs[pixel_index] = optix_launch_params.reservoirs[pixel_index];
        return;
    }
    float3 pos = make_float3(pos_with_flag);

    cuda::Random random;
    random.Init(4, pixel_index, optix_launch_params.random_seed);

    auto normal_depth = optix_launch_params.normal_buffer[pixel_index];
    auto normal = make_float3(normal_depth);
    auto albedo_flag = optix_launch_params.albedo_buffer[pixel_index];
    if (albedo_flag.w > 0.f) {
        optix_launch_params.final_reservoirs[pixel_index] = optix_launch_params.reservoirs[pixel_index];
        return;
    }

    float3 wo = optix::ToLocal(-normalize(pos - optix_launch_params.camera.pos), normal);

    auto albedo = make_float3(albedo_flag);

    Reservoir reservoir;
    reservoir.Init();
    unsigned int M = 0;

    // 取5个neighbour
    for (auto i = 0u; i < 5; ++i) {
        float r = optix_launch_params.spatial_radius * random.Next();
        float theta = M_PIf * 2.f * random.Next();
        // 随机一个方向的neighbour pixel
        int2 neighbor_pixel = make_int2(index.x + r * cos(theta), index.y + r * sin(theta));
        if (neighbor_pixel.x < 0 || neighbor_pixel.x >= w || neighbor_pixel.y < 0 || neighbor_pixel.y >= h)
            continue;
        const unsigned int neighbor_pixel_index = neighbor_pixel.y * w + neighbor_pixel.x;

        // 去掉一些特殊情况
        auto neighbor_normal_depth = optix_launch_params.normal_buffer[neighbor_pixel_index];
        if (dot(normal, make_float3(neighbor_normal_depth)) < 0.906307787f)
            continue;
        if (normal_depth.w * 0.9f > neighbor_normal_depth.w || normal_depth.w * 1.1f < neighbor_normal_depth.w)
            continue;

        // 取出neighbour reservoir中的样本，采样的光源点还是neighbour中的，但方位和材质都用自己的
        auto &neighbor_reservoir = optix_launch_params.reservoirs[neighbor_pixel_index];
        float3 wi = optix::ToLocal(normalize(neighbor_reservoir.y.pos - pos), normal);

        float3 f = make_float3(0.f);
        if (wi.z > 0.f && wo.z > 0.f) {
            f = albedo * M_1_PIf;
        }

        Reservoir::Sample x_i = neighbor_reservoir.y;
        x_i.radiance = make_float3(0.f);
        x_i.p_hat = 0.f;
        if (!optix::IsZero(f)) {
            x_i.radiance = x_i.emission * f * wi.z; // 原本是 x_i.radiance += emitter_sample_record.radiance * f * NoL;
            // phat
            auto &emitter = optix_launch_params.emitters.SelectOneEmiiter(x_i.emitter_rand.x);
            optix::EmitterSampleRecord emitter_sample_record;
            optix::LocalGeometry geo;
            geo.position = pos;
            geo.normal = normal;
            emitter.SampleDirect(emitter_sample_record, geo, make_float2(x_i.emitter_rand.y, x_i.emitter_rand.z));
            float NoL = dot(geo.normal, emitter_sample_record.wi);
            if(emitter_sample_record.pdf < 1e-5)
                emitter_sample_record.radiance = make_float3(0.0f);
            x_i.radiance += emitter_sample_record.radiance * f * NoL;
            x_i.p_hat = optix::GetLuminance(emitter_sample_record.radiance);
        }

        if(x_i.p_hat > 0){
            float w_i = x_i.p_hat * neighbor_reservoir.M * neighbor_reservoir.W;
            reservoir.Update(x_i, w_i, random); // 五个neighbour update五次
            M += neighbor_reservoir.M - 1;  
        }
    }
    reservoir.M = M; // 五个neighbour的M累加
    optix_launch_params.final_reservoirs[pixel_index] = optix_launch_params.reservoirs[pixel_index];
    reservoir.CalcW();
    if (reservoir.W > 0.f) {
        optix_launch_params.final_reservoirs[pixel_index].Combine(reservoir, random); // 和原本的reservoir合并
    }
    return;
}

extern "C" __global__ void __miss__default() {
}
extern "C" __global__ void __closesthit__default() {
}