#include "hip/hip_runtime.h"
#include "type.h"
#include <optix.h>

//#include "../indirect/indirect.h"
#include "../indirect/probemath.h"
#include "render/geometry.h"
#include "render/emitter.h"
#include "optix/util.h"

#include "cuda/random.h"

using namespace Pupil;

extern "C" {
__constant__ ddgi::shading::OptixLaunchParams optix_launch_params;
}

// uniform
__device__ int3 getBaseGridCoord(float3 probeStartPosition, float3 probeStep, int3 probeCount, float3 X) {
    return clamp(make_int3((X - probeStartPosition) / probeStep), make_int3(0, 0, 0), probeCount - make_int3(1));
}

// uniform
__device__ int gridCoordToProbeIndex(int3 probeCount, int3 probeCoords) {
    return int(probeCoords.x + probeCoords.y * probeCount.x + probeCoords.z * probeCount.x * probeCount.y);
    // return int(probeCoords.x * probeCount.y * probeCount.z + probeCoords.y * probeCount.z + probeCoords.z);
}

// uniform
__device__ float3 gridCoordToPosition(float3 probeStartPosition, float3 probeStep, int3 c) {
    return probeStep * make_float3(c) + probeStartPosition;
}

__device__ float4 bilinearInterpolation(const float4 *textureData,float2 texCoord, int fullTextureWidth, int fullTextureHeight){
    
    int x0 = floor(texCoord.x);
    int y0 = floor(texCoord.y);
    int x1 = x0 + 1;
    int y1 = y0 + 1;
    float u = texCoord.x - x0;
    float v = texCoord.y - y0;

    x0 = clamp(x0, 0, fullTextureWidth - 1);
    y0 = clamp(y0, 0, fullTextureHeight - 1);
    x1 = clamp(x1, 0, fullTextureWidth - 1);
    y1 = clamp(y1, 0, fullTextureHeight - 1);

    int index00 = y0 * fullTextureWidth + x0;
    int index01 = y0 * fullTextureWidth + x1;
    int index10 = y1 * fullTextureWidth + x0;
    int index11 = y1 * fullTextureWidth + x1;

    float4 pixel00 = textureData[index00];
    float4 pixel01 = textureData[index01];
    float4 pixel10 = textureData[index10];
    float4 pixel11 = textureData[index11];

    float4 result = (1 - u) * (1 - v) * pixel00 +
                u * (1 - v) * pixel01 +
                (1 - u) * v * pixel10 +
                u * v * pixel11;
    return result;

}

__device__ float2 textureCoordFromDirection(float3 dir, int probeIndex, int fullTextureWidth, int fullTextureHeight,
                                          int probeSideLength) {
    float2 normalizedOctCoord = octEncode(normalize(dir));
    float2 normalizedOctCoordZeroOne = (normalizedOctCoord + make_float2(1.0f)) * 0.5f;

    // Length of a probe side, plus one pixel on each edge for the border
    float probeWithBorderSide = (float)probeSideLength + 2.0f;

    float2 octCoordNormalizedToTextureDimensions = normalizedOctCoordZeroOne * (float)probeSideLength;
    int probesPerRow = (fullTextureWidth - 2) / (int)probeWithBorderSide;

    // Add (2,2) back to texCoord within larger texture. Compensates for 1 pix
    // border around texture and further 1 pix border around top left probe.
    float2 probeTopLeftPosition = make_float2((probeIndex % probesPerRow) * probeWithBorderSide,
                                              (probeIndex / probesPerRow) * probeWithBorderSide) +
                                  make_float2(2.0f);

    return probeTopLeftPosition + octCoordNormalizedToTextureDimensions;
}

__device__ float3 ComputeIndirect(const float3 wsN, const float3 wsPosition, const float3 rayorigin,
                                  const float4 *probeirradiance, const float4 *probedepth, float3 probeStartPosition,
                                  float3 probeStep, int3 probeCount, uint2 probeirradiancesize, int probeSideLength,
                                  float energyConservation) {

    const float epsilon = 1e-6;
    // gbuffer_WS_NORMAL_buffer
    // gbuffer_WS_POSITION_buffer
    // gbuffer_WS_RAY_ORIGIN_buffer
    // probe irradiance buffer

    if (dot(wsN, wsN) < 0.01) {
        return make_float3(0.0f);
    }

    int3 baseGridCoord = getBaseGridCoord(probeStartPosition, probeStep, probeCount, wsPosition);
    float3 baseProbePos = gridCoordToPosition(probeStartPosition, probeStep, baseGridCoord);

    float3 sumIrradiance = make_float3(0.0f);
    float sumWeight = 0.0f;

    //  alpha is how far from the floor(currentVertex) position. on [0, 1] for each axis.
    float3 alpha = clamp((wsPosition - baseProbePos) / probeStep, make_float3(0), make_float3(1));

    for (int i = 0; i < 8; ++i) {
        float weight = 1.0;
        int3 offset = make_int3(i & 1, (i >> 1) & 1, (i >> 2) & 1);
        int3 probeGridCoord = clamp(baseGridCoord + offset, make_int3(0), probeCount - make_int3(1));
        int probeIndex = gridCoordToProbeIndex(probeCount, probeGridCoord);
        float3 probePos = gridCoordToPosition(probeStartPosition, probeStep, probeGridCoord);
        
        // Smooth backface test
        {
            float3 trueDirectionToProbe = normalize(probePos - wsPosition);
            // weight *= max(0.0001, dot(trueDirectionToProbe, wsN));
            weight *= pow(max(0.0001f, (dot(trueDirectionToProbe, wsN) + 1.0) * 0.5), 2) + 0.2;
        }

        // Moment visibility test (chebyshev)
        {
            float normalBias = 0.05f;
            float3 w_o = normalize(rayorigin - wsPosition);
            float3 probeToPoint = wsPosition - probePos + (wsN + 3.0 * w_o) * normalBias;
     
            float3 dir = normalize(-probeToPoint);
            float2 texCoord = textureCoordFromDirection(-dir, probeIndex, probeirradiancesize.x, probeirradiancesize.y,
                                                      probeSideLength);
            // float4 temp = probedepth[texCoord.x + texCoord.y * probeirradiancesize.x];
            float4 temp = bilinearInterpolation(probedepth, texCoord, probeirradiancesize.x, probeirradiancesize.y);
            float mean = temp.x;
            float variance = abs(pow(temp.x, 2) - temp.y);

            float distToProbe = length(probeToPoint);
            float chebyshevWeight = variance / (variance + pow(max(distToProbe - mean, 0.0), 2));
            chebyshevWeight = max(pow(chebyshevWeight, 3), 0.0f);

            weight *= (distToProbe <= mean) ? 1.0 : chebyshevWeight;
        }

        // Avoid zero
        weight = max(0.000001, weight);

        const float crushThreshold = 0.2;
        if (weight < crushThreshold) {
            weight *= weight * weight * (1.0 / pow(crushThreshold, 2));
        }

        // Trilinear
        float3 trilinear = (1.0 - alpha) * (1 - make_float3(offset)) + alpha * make_float3(offset);
        weight *= trilinear.x * trilinear.y * trilinear.z;

        float2 texCoord = textureCoordFromDirection(normalize(wsN), probeIndex, probeirradiancesize.x,
                                                  probeirradiancesize.y, probeSideLength);

        float4 irradiance = bilinearInterpolation(probeirradiance, texCoord, probeirradiancesize.x, probeirradiancesize.y);
        sumIrradiance += weight * make_float3(irradiance.x, irradiance.y, irradiance.z);
        sumWeight += weight;
    }

    float3 netIrradiance = sumIrradiance / sumWeight;
    netIrradiance *= energyConservation;
    float3 indirect = 2.0 * M_PIf * netIrradiance;

    return indirect;
}

extern "C" __global__ void __raygen__main() {
    const uint3 index = optixGetLaunchIndex();
    const unsigned int w = optix_launch_params.config.frame.width;
    const unsigned int h = optix_launch_params.config.frame.height;
    const unsigned int pixel_index = index.y * w + index.x;

    const auto pos = optix_launch_params.position_buffer[pixel_index];
    const auto albedo = optix_launch_params.albedo_buffer[pixel_index];

    float3 direct_color = make_float3(0.f);
    if(optix_launch_params.directOn){
        
        if (pos.w > 0.f) {
            
            if (albedo.w > 0.f) {
                direct_color = make_float3(albedo);
            } else {
                float3 hit_pos = make_float3(pos);
                float3 hit_nor = make_float3(optix_launch_params.normal_buffer[pixel_index]);
                auto &reservoir = optix_launch_params.final_reservoirs[pixel_index];
                direct_color = reservoir.y.radiance * reservoir.W;
                // direct_color = reservoir.y.emission * reservoir.W;
                // direct_color = make_float3(reservoir.W);
                // if(optix::GetLuminance(reservoir.y.radiance) < 1e-5){
                //     direct_color = make_float3(1,1,0);
                // }
                // direct_color = reservoir.y.emission;
                //direct_color = reservoir.y.radiance;
            }

        } else {
            //TODO env light
        }

        
    }
    float3 indirect_color = make_float3(0.f);
    if(optix_launch_params.indirectOn && albedo.w <= 0.f && pos.w > 0.f){
        auto &camera = *optix_launch_params.camera.GetDataPtr();
        float3 camera_pos = make_float3(
            camera.camera_to_world.r0.w,
            camera.camera_to_world.r1.w,
            camera.camera_to_world.r2.w);

        float3 indirect_light = ComputeIndirect(make_float3(optix_launch_params.normal_buffer[pixel_index]),
                                                make_float3(pos.x,pos.y,pos.z),
                                                camera_pos,
                                                optix_launch_params.probeirradiance.GetDataPtr(),
                                                optix_launch_params.probedepth.GetDataPtr(),
                                                optix_launch_params.probeStartPosition,
                                                optix_launch_params.probeStep,
                                                optix_launch_params.probeCount,
                                                optix_launch_params.probeirradiancesize,
                                                optix_launch_params.probeSideLength,
                                                optix_launch_params.energyConservation);
        indirect_color = indirect_light * make_float3(albedo) * M_1_PIf;
    }


    optix_launch_params.frame_buffer[pixel_index] = make_float4(direct_color + indirect_color, 1.f);

}

extern "C" __global__ void __miss__default() {

}

extern "C" __global__ void __closesthit__default() {

}
