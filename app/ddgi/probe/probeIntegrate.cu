#include "hip/hip_runtime.h"
#include "cuda/vec_math.h"
#include "probeIntegrate.h"

__device__ float2 octEncode(float3 v)
{
    float l1norm = fabsf(v.x) + fabsf(v.y) + fabsf(v.z);
    float2 result = make_float2(v.x, v.y) * (1.0f / l1norm);
    if (v.z < 0.0f)
    {
        result = (make_float2(1.0f) - make_float2(fabsf(result.y), fabsf(result.x))) *
                 (make_float2(result.x >= 0 ? 1.0 : 0.0, result.y >= 0 ? 1.0 : 0.0));
    }
    return result;
}

__device__ float3 octDecode(float2 o)
{
    float3 v = make_float3(o.x, o.y, 1.0f - fabsf(o.x) - fabsf(o.y));
    if (v.z < 0.0f)
    {
        float2 xy = (make_float2(1.0f) - make_float2(fabsf(v.y), fabsf(v.x))) *
                    (make_float2(v.x >= 0 ? 1.0 : 0.0, v.y >= 0 ? 1.0 : 0.0));
        v.x = xy.x;
        v.y = xy.y;
    }
    return normalize(v);
}

__device__ float2 normalizedOctCoord(int2 fragCoord, int probeSideLength)
{
    int probeWithBorderSide = probeSideLength + 2;

    float2 octFragCoord = make_float2((fragCoord.x - 2) % probeWithBorderSide, (fragCoord.y - 2) % probeWithBorderSide);
    // Add back the half pixel to get pixel center normalized coordinates
    return (octFragCoord + make_float2(0.5)) * (2.0f / float(probeSideLength)) - make_float2(1.0f);
}

// probeRayGbuffer -> probeTexture
__global__ void UpdateProbe(const float4 *rayGbuffer, float4 *probeIrradiance, uint2 size, int raysPerProbe,
                            int probeSideLength)
{

    int pixel_x = threadIdx.x + blockIdx.x * blockDim.x;
    int pixel_y = threadIdx.y + blockIdx.y * blockDim.y;
    if (pixel_x >= size.x)
        return;
    if (pixel_y >= size.y)
        return;
    int pixel_index = pixel_x + size.x * pixel_y;

    for (int r = 0; r < raysPerProbe; ++r)
    {

        // 计算probeId
        int probeWithBorderSide = probeSideLength + 2;
        int probesPerSide = (size.x - 2) / probeWithBorderSide;
        int probeId = int(pixel_x / probeWithBorderSide) + probesPerSide * int(pixel_y / probeWithBorderSide);

        // 计算direction
        float3 texelDirection = octDecode(normalizedOctCoord(make_int2(pixel_x, pixel_y), probeSideLength));

        // hit radiance
        int2 coord = make_int2(r, probeId);
        float4 rayHitRadiance = probeIrradiance[raysPerProbe * coord.y + coord.x];

        // float weight = max(0.0, dot(texelDirection, rayDirection));
    }

    probeIrradiance[pixel_index] = make_float4(0.5);
}

// void UpdateProbeCPU(hipStream_t stream, Pupil::cuda::ConstArrayView<float4> rayGbuffer,
//                     Pupil::cuda::RWArrayView<float4> &probeIrradiance, uint2 size, int raysPerProbe,
//                     int probeSideLength)
void UpdateProbeCPU(hipStream_t stream, Pupil::ddgi::probe::UpdateParams update_params, uint2 size, int raysPerProbe,
                    int probeSideLength)
{

    constexpr int block_size_x = 32;
    constexpr int block_size_y = 32;
    int grid_size_x = (size.x + block_size_x - 1) / block_size_x;
    int grid_size_y = (size.y + block_size_y - 1) / block_size_y;
    UpdateProbe<<<dim3(grid_size_x, grid_size_y), dim3(block_size_x, block_size_y), 0, stream>>>(
        update_params.rayradiance.GetDataPtr(), update_params.probeirradiance.GetDataPtr(), size, raysPerProbe,
        probeSideLength);
    // UpdateProbe<<<dim3(grid_size_x, grid_size_y), dim3(block_size_x, block_size_y), 0, stream>>>(
    //     rayGbuffer.GetDataPtr(), probeIrradiance.GetDataPtr(), size, raysPerProbe, probeSideLength);
}
